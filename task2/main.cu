#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#define N 1024  
//#define BLOCK_SIZE 8
//#define BLOCK_SIZE 16
//#define BLOCK_SIZE 32  
//#define BLOCK_SIZE 64
//#define BLOCK_SIZE 128
#define BLOCK_SIZE 256


// Ядро для блочного умножения матриц(Матрицы A и B разделяются на блоки размером BLOCK_SIZE x BLOCK_SIZE)
__global__ void blockMatrixMulKernel(float *A, float *B, float *C, int width) {
    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];
    // Каждый поток вычисляет элемент результата C в блоке размером BLOCK_SIZE x BLOCK_SIZE.
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float sum = 0.0f;

    // Разбиваем вычисления на блоки
    for (int i = 0; i < width / BLOCK_SIZE; ++i) {
        
        sharedA[threadIdx.y][threadIdx.x] = A[row * width + (i * BLOCK_SIZE + threadIdx.x)];
        sharedB[threadIdx.y][threadIdx.x] = B[(i * BLOCK_SIZE + threadIdx.y) * width + col];

        __syncthreads();  

        // Умножаем блоки
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            sum += sharedA[threadIdx.y][j] * sharedB[j][threadIdx.x];
        }

        __syncthreads();  
    }

    
    if (row < width && col < width) {
        C[row * width + col] = sum;
    }
}


void matrixMul(float *A, float *B, float *C, int width) {
    float *d_A, *d_B, *d_C;
    size_t size = width * width * sizeof(float);

    
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((width + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (width + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    
    blockMatrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, width);

   
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int width = N;
    float *A = (float*)malloc(width * width * sizeof(float));
    float *B = (float*)malloc(width * width * sizeof(float));
    float *C = (float*)malloc(width * width * sizeof(float));

   
    for (int i = 0; i < width * width; ++i) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

   
    clock_t start = clock();
    matrixMul(A, B, C, width);
    clock_t end = clock();
    
    
    printf("Time taken: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Освобождаем память
    free(A);
    free(B);
    free(C);

    return 0;
}
